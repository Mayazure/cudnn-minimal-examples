// $env:CUDNN_LOGINFO_DBG=1
// $env:CUDNN_LOG_ERR=1
// $env:CUDNN_LOGWARN_DBG=1
// $env:CUDNN_LOGDEST_DBG="stdout"
// $env:CUDNN_LOGERR_DBG=1

// nvcc -I "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.7\include\nvtx3" -I C:\Users\mayaz\Programs\cudnn-windows-x86_64-8.6.0.163_cuda11-archive\include -L C:\Users\mayaz\Programs\cudnn-windows-x86_64-8.6.0.163_cuda11-archive\lib\x64 -l cudnn -l cudart -l cublas -l cublasLt -o testcudnn_convdatabwd .\testcudnn_convdatabwd.cu
// testcudnn_convdatabwd.cu

#include <iostream>
#include <hip/hip_runtime.h>
#include "hipDNN.h"
// #include "nvToolsExt.h"

int main(int argc, char** argv)
{    
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::cout << "numGPUs: " << numGPUs << std::endl;
    hipSetDevice(1); // use GPU0
    int device; 
    struct hipDeviceProp_t devProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&devProp, device);
    std::cout << "GPU0: " << devProp.name << std::endl;
    std::cout << "Total global mem: " << devProp.totalGlobalMem/1024/1024/1024 << " GB" << std::endl;
    std::cout << "Total const mem: " << devProp.totalConstMem/1024/1024/1024 << " GB" << std::endl;
    std::cout << "Compute capability:" << devProp.major << "." << devProp.minor << std::endl;

    hipdnnHandle_t handle_;
    hipdnnCreate(&handle_);
    std::cout << "Created cuDNN handle" << std::endl;

    hipdnnStatus_t cudnnStatus;
    cudnnStatus = cudnnCnnInferVersionCheck();
    std::cout << "Cudnn Status: " << cudnnStatus << std::endl;

    float alpha[1] = {1};
    float beta[1] = {0.0};
    
    // Create filter
    hipdnnFilterDescriptor_t wDesc;
    hipdnnStatus_t status = hipdnnCreateFilterDescriptor(&wDesc);
    status = hipdnnSetFilter4dDescriptor(
        wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 
        2, 2, 3, 3
    );
    
    float *w_host = (float*)malloc(18*sizeof(float));
    for(int i=0;i<9;i++){
        w_host[i] = 1.0f;
    }
    for(int i=9;i<18;i++){
        w_host[i] = 2.0f;
    }
    float *w;
    hipMalloc(&w, 18 * sizeof(float));
    hipMemcpy(w, w_host, 18*sizeof(float), hipMemcpyHostToDevice);

    // Create tensor
    hipdnnTensorDescriptor_t dyDesc;
    status = hipdnnCreateTensorDescriptor(&dyDesc);
    status = hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 2, 2, 2);
    float *dy_host = (float*)malloc(8*sizeof(float));
    for(int i=0;i<8;i++){
        dy_host[i] = 1.0f;
    }
    float *dy;
    hipMalloc(&dy, 8 * sizeof(float));
    hipMemcpy(dy, dy_host, 8*sizeof(float), hipMemcpyHostToDevice);

    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);
    hipdnnSetConvolutionGroupCount(convDesc, 2);

    // Create dx
    hipdnnTensorDescriptor_t dxDesc;
    hipdnnCreateTensorDescriptor(&dxDesc);
    hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 2, 4, 4);
    float *dx_host = (float*)malloc( 32 * sizeof(float) );
    for(int i=0;i<32;i++) {
        dx_host[i] = (i+1) * 1.0f;
    }
    float *dx;
    hipMalloc(&dx, 32 * sizeof(float));
    hipMemset((void**)&dx, 0, 32*sizeof(float));

    hipdnnConvolutionBwdDataAlgo_t algos[] = {
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD_NONFUSED
    };

    const char* algoNames[6] = {
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0",
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1",
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT",
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING",
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD",
        "HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD_NONFUSED"
    };

    for(int i=0;i<6;i++){
        std::cout<<"--------"<<std::endl<<algoNames[i]<<std::endl;
        hipdnnConvolutionBwdDataAlgo_t algo = algos[i];

        size_t workspaceSizeInBytes;
        status = hipdnnGetConvolutionBackwardDataWorkspaceSize(
                    handle_,
                    wDesc,
                    dyDesc,
                    convDesc,
                    dxDesc,
                    algo,
                    &workspaceSizeInBytes
                );
        std::cout<<"WorkspaceSize: "<<workspaceSizeInBytes<<std::endl;
        void *workspace;
        hipMalloc(&workspace, workspaceSizeInBytes);

        cudnnStatus = hipdnnConvolutionBackwardData(
            handle_,
            alpha,
            wDesc,
            w,
            dyDesc,
            dy,
            convDesc,
            algo,
            workspace,
            workspaceSizeInBytes,
            beta,
            dxDesc,
            dx
        );

        hipMemcpy( dx_host, dx, 32 * sizeof(float), hipMemcpyDeviceToHost);
        std::cout<<"dx:";
        for (int i=0;i<32;i++){
                if(i%4==0){
                    std::cout << std::endl;
                };
            std::cout << dx_host[i] << " ";
        };
        std::cout << std::endl;
        hipFree(workspace);
    }

    hipFree(w);
    hipFree(dy);
    hipFree(dx);
    
    free(dx_host);
    free(dy_host);
    free(w_host);

    hipdnnDestroy(handle_);
    return 0;
}